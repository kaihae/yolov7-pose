#include "hip/hip_runtime.h"
#include "yololayer.h"

namespace Tn
{
    template<typename T> 
    void write(char*& buffer, const T& val)
    {
        *reinterpret_cast<T*>(buffer) = val;
        buffer += sizeof(T);
    }

    template<typename T> 
    void read(const char*& buffer, T& val)
    {
        val = *reinterpret_cast<const T*>(buffer);
        buffer += sizeof(T);
    }
}

#define CUDA_CHECK(callstr)\
    {\
        hipError_t error_code = callstr;\
        if (error_code != hipSuccess) {\
            std::cerr << "CUDA error " << error_code << " at " << __FILE__ << ":" << __LINE__;\
            assert(0);\
        }\
    }

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int classCount, int netWidth, int netHeight, int maxOut, const std::vector<Yolo::YoloKernel>& vYoloKernel)
    {
        mClassCount = classCount;
        mYoloV5NetWidth = netWidth;
        mYoloV5NetHeight = netHeight;
        mMaxOutObject = maxOut;
        mYoloKernel = vYoloKernel;
        mKernelCount = vYoloKernel.size();

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    YoloLayerPlugin::~YoloLayerPlugin()
    {
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipFree(mAnchor[ii]));
        }
        CUDA_CHECK(hipHostFree(mAnchor));
    }

    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        read(d, mYoloV5NetWidth);
        read(d, mYoloV5NetHeight);
        read(d, mMaxOutObject);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(mYoloKernel.data(), d, kernelSize);
        d += kernelSize;
        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT * 2;
        for (int ii = 0; ii < mKernelCount; ii++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii], AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer) const TRT_NOEXCEPT
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        write(d, mYoloV5NetWidth);
        write(d, mYoloV5NetHeight);
        write(d, mMaxOutObject);
        auto kernelSize = mKernelCount * sizeof(YoloKernel);
        memcpy(d, mYoloKernel.data(), kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }

    size_t YoloLayerPlugin::getSerializationSize() const TRT_NOEXCEPT
    {
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size() + sizeof(mYoloV5NetWidth) + sizeof(mYoloV5NetHeight) + sizeof(mMaxOutObject);
    }

    int YoloLayerPlugin::initialize() TRT_NOEXCEPT
    {
        return 0;
    }

    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) TRT_NOEXCEPT
    {
        //output the result to channel
        int totalsize = mMaxOutObject * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void YoloLayerPlugin::setPluginNamespace(const char* pluginNamespace) TRT_NOEXCEPT
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloLayerPlugin::getPluginNamespace() const TRT_NOEXCEPT
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const TRT_NOEXCEPT
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const TRT_NOEXCEPT
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const TRT_NOEXCEPT
    {
        return false;
    }

    void YoloLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput) TRT_NOEXCEPT
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator) TRT_NOEXCEPT
    {
    }

    // Detach the plugin object from its execution context.
    void YoloLayerPlugin::detachFromContext() TRT_NOEXCEPT {}

    const char* YoloLayerPlugin::getPluginType() const TRT_NOEXCEPT
    {
        return "YoloLayer_TRT";
    }

    const char* YoloLayerPlugin::getPluginVersion() const TRT_NOEXCEPT
    {
        return "1";
    }

    void YoloLayerPlugin::destroy() TRT_NOEXCEPT
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloLayerPlugin::clone() const TRT_NOEXCEPT
    {
        YoloLayerPlugin* p = new YoloLayerPlugin(mClassCount, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, mYoloKernel);
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data) { return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output, int noElements,
        const int netwidth, const int netheight, int maxoutobject, int yoloWidth, int yoloHeight, const float anchors[CHECK_COUNT * 2], int classes, int outputElem)
    {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid * bnIdx;
        int info_len_i = 5 + classes;
        int info_len_kpt = KEY_POINTS_NUM * 3;
        const float* curInput = input + bnIdx * ((info_len_i + info_len_kpt) * total_grid * CHECK_COUNT);

        for (int k = 0; k < CHECK_COUNT; ++k) {
            float box_prob = Logist(curInput[idx + k * (info_len_i + info_len_kpt) * total_grid + 4 * total_grid]);
            if (box_prob < IGNORE_THRESH) continue;
            int class_id = 0; //person class
            // float max_cls_prob = 0.0;
            // for (int i = 5; i < info_len_i; ++i) {
            //     float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
            //     if (p > max_cls_prob) {
            //         max_cls_prob = p;
            //         class_id = i - 5;
            //     }
            // }
            float max_cls_prob = Logist(curInput[idx + k * (info_len_i + info_len_kpt) * total_grid + 5 * total_grid]);

            float *res_count = output + bnIdx * outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= maxoutobject) return;
            char *data = (char*)res_count + sizeof(float) + count * sizeof(Detection);
            Detection *det = (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //------------bboxs------------
            //Location
            // pytorch:
            // y = x[i].sigmoid()
            // xy = (y[..., 0:2] * 2. - 0.5 + self.grid[i]) * self.stride[i]  # xy
            // wh = (y[..., 2:4] * 2) ** 2 * self.anchor_grid[i].view(1, self.na, 1, 1, 2) # wh
            
            det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * (info_len_i + info_len_kpt) * total_grid + 0 * total_grid])) * netwidth / yoloWidth;
            det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * (info_len_i + info_len_kpt) * total_grid + 1 * total_grid])) * netheight / yoloHeight;

            // W: (Pw * e^tw) / FeaturemapW * netwidth
            // v5: https://github.com/ultralytics/yolov5/issues/471
            det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2 * k];
            det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2 * k + 1];
            det->conf = box_prob * max_cls_prob;
            det->class_id = class_id;

            //------------keypoints------------
            // Location
            //pytorch:
            // x_kpt[..., 0::3] = (x_kpt[..., ::3] * 2. - 0.5 + kpt_grid_x.repeat(1,1,1,1,17)) * self.stride[i]  # xy
            // x_kpt[..., 1::3] = (x_kpt[..., 1::3] * 2. - 0.5 + kpt_grid_y.repeat(1,1,1,1,17)) * self.stride[i]  # xy
            // x_kpt[..., 2::3] = x_kpt[..., 2::3].sigmoid()
            for (int kpt_idx = 0; kpt_idx < KEY_POINTS_NUM; ++kpt_idx)
            {
                det->kpts[kpt_idx].x = (col - 0.5f + 2.0f * (curInput[idx + k * (info_len_i + info_len_kpt) * total_grid + (6 + kpt_idx*3) * total_grid]) ) * netwidth / yoloWidth;
                det->kpts[kpt_idx].y = (row - 0.5f + 2.0f * (curInput[idx + k * (info_len_i + info_len_kpt) * total_grid + (7 + kpt_idx*3) * total_grid]) ) * netheight / yoloHeight;
                det->kpts[kpt_idx].kpt_conf = Logist(curInput[idx + k * (info_len_i + info_len_kpt) * total_grid + (8 + kpt_idx*3) * total_grid]);                
            }

        }
    }

    void YoloLayerPlugin::forwardGpu(const float* const* inputs, float *output, hipStream_t stream, int batchSize)
    {
        int outputElem = 1 + mMaxOutObject * sizeof(Detection) / sizeof(float);
        for (int idx = 0; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemsetAsync(output + idx * outputElem, 0, sizeof(float), stream));
        }
        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i) {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width * yolo.height * batchSize;
            if (numElem < mThreadCount) mThreadCount = numElem;

            //printf("Net: %d  %d \n", mYoloV5NetWidth, mYoloV5NetHeight);
            CalDetection << < (numElem + mThreadCount - 1) / mThreadCount, mThreadCount, 0, stream >> >
                (inputs[i], output, numElem, mYoloV5NetWidth, mYoloV5NetHeight, mMaxOutObject, yolo.width, yolo.height, (float*)mAnchor[i], mClassCount, outputElem);
        }
    }


    int YoloLayerPlugin::enqueue(int batchSize, const void* const* inputs, void* TRT_CONST_ENQUEUE* outputs, void* workspace, hipStream_t stream) TRT_NOEXCEPT
    {
        forwardGpu((const float* const*)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection YoloPluginCreator::mFC{};
    std::vector<PluginField> YoloPluginCreator::mPluginAttributes;

    YoloPluginCreator::YoloPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloPluginCreator::getPluginName() const TRT_NOEXCEPT
    {
        return "YoloLayer_TRT";
    }

    const char* YoloPluginCreator::getPluginVersion() const TRT_NOEXCEPT
    {
        return "1";
    }

    const PluginFieldCollection* YoloPluginCreator::getFieldNames() TRT_NOEXCEPT
    {
        return &mFC;
    }

    IPluginV2IOExt* YoloPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) TRT_NOEXCEPT
    {
        // int class_count = 1;//p_netinfo[0];
        // int input_w = 960;//p_netinfo[1];
        // int input_h = 960;//p_netinfo[2];
        // int max_output_object_count = 1000;//p_netinfo[3];
        
        std::vector<Yolo::YoloKernel> kernels{Yolo::yolo4, Yolo::yolo3, Yolo::yolo2, Yolo::yolo1};
        
        YoloLayerPlugin* obj = new YoloLayerPlugin(CLASS_NUM, INPUT_W, INPUT_H, MAX_OUTPUT_BBOX_COUNT, kernels);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength) TRT_NOEXCEPT
    {
        // This object will be deleted when the network is destroyed, which will
        // call YoloLayerPlugin::destroy()
        YoloLayerPlugin* obj = new YoloLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
}